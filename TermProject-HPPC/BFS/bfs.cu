#include "hip/hip_runtime.h"
#include "driver.h"
#include "bfs.h"
#include "cuda_utils.h"
#include "assert.h"
#include <sys/time.h>
#include "omp.h"

#define BLOCK_SIZE 1024
#define FTS 4
#define TILE_SIZE 8
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

//#define DEBUG
//#define CUDADEBUG

#ifdef DEBUG
#define dbgprint1(a1)                           fprintf(stderr,a1)
#define dbgprint2(a1,a2)                        fprintf(stderr,a1,a2)
#define dbgprint3(a1,a2,a3)                     fprintf(stderr,a1,a2,a3)
#define dbgprint4(a1,a2,a3,a4)                  fprintf(stderr,a1,a2,a3,a4)
#define dbgprint5(a1,a2,a3,a4,a5)               fprintf(stderr,a1,a2,a3,a4,a5)
#define dbgprint6(a1,a2,a3,a4,a5,a6)            fprintf(stderr,a1,a2,a3,a4,a5,a6)
#define dbgprint7(a1,a2,a3,a4,a5,a6,a7)         fprintf(stderr,a1,a2,a3,a4,a5,a6,a7)
#define dbgprint8(a1,a2,a3,a4,a5,a6,a7,a8)      fprintf(stderr,a1,a2,a3,a4,a5,a6,a7,a8)

#else
#define dbgprint1(a1)
#define dbgprint2(a1,a2)
#define dbgprint3(a1,a2,a3)
#define dbgprint4(a1,a2,a3,a4)
#define dbgprint5(a1,a2,a3,a4,a5)
#define dbgprint6(a1,a2,a3,a4,a5,a6)
#define dbgprint7(a1,a2,a3,a4,a5,a6,a7)
#define dbgprint8(a1,a2,a3,a4,a5,a6,a7,a8)
#endif

#ifdef CUDADEBUG
#define cudadbgprint1(a1)                           printf(a1)
#define cudadbgprint2(a1,a2)                        printf(a1,a2)
#define cudadbgprint3(a1,a2,a3)                     printf(a1,a2,a3)
#define cudadbgprint4(a1,a2,a3,a4)                  printf(a1,a2,a3,a4)
#define cudadbgprint5(a1,a2,a3,a4,a5)               printf(a1,a2,a3,a4,a5)
#define cudadbgprint10(a1,a2,a3,a4,a5,a6,a7,a8,a9,a10)               printf(a1,a2,a3,a4,a5,a6,a7,a8,a9,a10)

#else
#define cudadbgprint1(a1)
#define cudadbgprint2(a1,a2)
#define cudadbgprint3(a1,a2,a3)
#define cudadbgprint4(a1,a2,a3,a4)
#define cudadbgprint5(a1,a2,a3,a4,a5)
#define cudadbgprint10(a1,a2,a3,a4,a5,a6,a7,a8,a9,a10)
#endif

void
printArray (dtype *A, int N) { 
	int i;

	for (i = 0; i < N ; i++) {
		dbgprint2("%u ", A[i]);
	}
	dbgprint1("\n");
}

void
printIntArray (int *A, int N) {
	int i;

	for (i = 0; i < N ; i++) {
		dbgprint2("%d ", A[i]);
	}
        dbgprint1("\n");
}

/* Each thread checks one edge of the edgelist and sets the depth of second vertex if the the first
 * vertex has a valid depth.
 * 'done' variable is a flag to indicate the completion of the search.
 */
__global__
void
BFSNaiveKernel (dtype *E, int *D, int current_depth, unsigned int num_edges, bool *done, int nBlocks)
{
	int tidx = ((blockIdx.y) * blockDim.x) + (blockIdx.x * nBlocks * blockDim.x) + threadIdx.x;

	if (tidx < num_edges) {
		dtype vfirst = E[2 * tidx];
		dtype vsecond = E[2 * tidx + 1];
		int dfirst = D[vfirst];
		int dsecond = D[vsecond];

		if ((dfirst == current_depth) && (dsecond == -1)) {
			D[vsecond] = dfirst + 1;
			*done = false;
		}
	}
}

/* Naive implementation of BFS using Edgelist representation of the graph. */
void
BFSNaive (dtype* E, unsigned int N, unsigned int num_edges, int* D, int* h_D)
{
	unsigned int nBlocks;
	int current_depth = 0;
	bool h_true = true, h_done = false, *d_done = NULL;

	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_done, sizeof (bool)));

	nBlocks = (num_edges + BLOCK_SIZE - 1) / (BLOCK_SIZE);
	nBlocks = sqrt(nBlocks);
	dim3 block(BLOCK_SIZE,1);
	dim3 grid(nBlocks,nBlocks);
        while(!h_done) {
		CUDA_CHECK_ERROR (hipMemcpy(d_done, &h_true, sizeof(bool), hipMemcpyHostToDevice));
                BFSNaiveKernel <<<grid,block>>>(E, D, current_depth, num_edges, d_done,nBlocks);
		current_depth++;
		CUDA_CHECK_ERROR (hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost));
        }
	CUDA_CHECK_ERROR (hipFree (d_done));
}

/* Naive implementation of BFS using Edgelist representation of the graph.
 *
 * This method iterates multiple times over the whole edgelist setting the depths of the
 * vertices at the current level, till all the vertices has a valid depth.
 */
void
cudaBFS_Edgelist (dtype* h_E, unsigned int N, unsigned int num_edges, int *h_D)
{
	dtype *d_E;
	int *d_D;
	
	struct timeval start, end;
	gettimeofday(&start, NULL);

	h_D[0] = 0;
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_E, (num_edges * 2) * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_E, h_E, (num_edges * 2) * sizeof (dtype), hipMemcpyHostToDevice));

	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_D, N * sizeof (int)));
	CUDA_CHECK_ERROR (hipMemcpy (d_D, h_D, N * sizeof (int), hipMemcpyHostToDevice));
	
	BFSNaive (d_E, N, num_edges, d_D, h_D);

	gettimeofday(&end, NULL);

	CUDA_CHECK_ERROR (hipMemcpy (h_D, d_D, N * sizeof (int), hipMemcpyDeviceToHost));

	fprintf (stderr, "Time to execute GPU BFS Edgelist code: %ld usecs\n", ((end.tv_sec * 1000000 + end.tv_usec)
										  - (start.tv_sec * 1000000 + start.tv_usec)));
	CUDA_CHECK_ERROR (hipFree (d_E));
	CUDA_CHECK_ERROR (hipFree (d_D));
}

/* Input: F (Present frontier), N (Frontier size), R, R_size
 * Output: AllocationSize (Offsets of starting neighbours in next frontier)
 * 
 * Load the number of neighbours of each vertex 'v' in F, using R[v+1]-R[v]
 * into shared memory and output the inclusive prefix sum into Allocation Size
 * sum contains the last element after the prefix sum - this is used to
 * propagate the sum to other blocks for completing the prefix sum
 *
 * Each thread loads two elements of the input to prefix sum
 * Shared memory is two times the frontier size since we try to load the input
 * into conflict free indices in the shared memory to avoid shared memory bank
 * conflicts
 */
__global__ void prefixscan (int *AllocationSize, const dtype *F, int N, int tbSize, dtype *R, int R_size, int *sum)
{
	__shared__ int temp[4 * BLOCK_SIZE];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nthreads = (N + 1) /2;
	int nblocks = (nthreads + tbSize - 1) / tbSize;
	int id = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	int offset = 1;
	int n = 0;

	if (bid == nblocks - 1) {
		n = N - ((nblocks - 1) * 2 * tbSize);
	} else {
		n = 2 * tbSize;
	}
	__syncthreads();
	int ai_in = id;
	int bi_in = id + (n/2);
	int ai = tid;
	int bi = tid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	int d;
	
	if (bi < n) {
		__syncthreads();
		if (ai < (n/2)) {
			dtype idx = F[ai_in];
			temp[ai + bankOffsetA] = R[idx + 1] - R[idx];
		}
		dtype idx = F[bi_in];
		temp[bi + bankOffsetB] = R[idx + 1] - R[idx];

		for (d = n >> 1; d > 0; d >>= 1) { 
			__syncthreads();
  			if (tid < d) {
				int ai = offset*(2*tid+1)-1;
				int bi = offset*(2*tid+2)-1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
	
				if (bi < n) {
					temp[bi] += temp[ai];
				}
	                }
			offset *= 2;
		}
  
		for (d = 1; d <= (n/2); d *= 2) {
     			offset >>= 1;
			int i = n >> 1, j = d*2;
			while (j <= i) {
				i >>= 1;
			}
	   		 __syncthreads();
   			 if (tid < i) {
     				 int ai = offset*(2*tid+2)-1;
     				 int bi = offset*(2*tid+3)-1;
	     			 ai += CONFLICT_FREE_OFFSET(ai);
      				 bi += CONFLICT_FREE_OFFSET(bi); 
				 if (bi < n) {
	        			 temp[bi] += temp[ai];
			 	}
     			}
		}
		__syncthreads();
		if (ai < (n/2)) {
			AllocationSize[ai_in] = temp[ai + bankOffsetA];
		}
		if (bi < n) {
			AllocationSize[bi_in] = temp[bi + bankOffsetB];
		}
		if (tid == 0) {
                        sum[bid] = temp[n - 1 + CONFLICT_FREE_OFFSET(n-1)];
                }
	}
}

/* Helper function for prefixscan2
 * Prefix sum on the last elements of the blocks obtained using prefixscan
 * or maskPrefixscan
 */
__global__ void sumPrefix (int *sum, int n) {
	for (int i = 1; i < n; i++) {
		sum[i] += sum[i-1];
	}
}

/* Complete the prefix sum by adding 'sum' element corr to prev block to
 * all the elements of a block
 */
__global__ void prefixscan2 (int *g_odata, int N, int tbSize, int *sum)
{
	int bid = blockIdx.x;
	int id = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
	int s = (bid == 0) ? 0 : sum[bid-1];

	if (id < N) {
		g_odata[id] += s;
		if (id + 1 < N)
			g_odata[id + 1] += s;
	}
}

/* Inclusive prefix scan
 * Input: g_idata (input vector), N (input size), tbSize (thread block size)
 * Output: g_odata (output vector after inclusive prefix sum)
 *
 * Each thread loads two elements of the input to prefix sum.
 * Shared memory is two times the frontier size since we try to load the input
 * into conflict free indices in the shared memory to avoid shared memory bank
 * conflicts. 'sum' contains the last element after the prefix sum - this is used to
 * propagate the sum to other blocks for completing the prefix sum.
 *
 * Similar to prefixscan but this is generic while prefixscan was specific to AllocationSize of the
 * frontier as it load the number of neighbors before performing the scan.
 */
__global__ void maskPrefixscan (int *g_odata, int *g_idata, int N, int tbSize, int *sum)
{
	__shared__ int temp[4 * BLOCK_SIZE];
        int tid = threadIdx.x;
        int bid = blockIdx.x;
        int nthreads = (N + 1) /2;
        int nblocks = (nthreads + tbSize - 1) / tbSize;
        int id = 2 * blockIdx.x * blockDim.x + threadIdx.x;
        int offset = 1;
        int n = 0;

        if (bid == nblocks - 1) {
                n = N - ((nblocks - 1) * 2 * tbSize);
        } else {
                n = 2 * tbSize;
        }

	int ai_in = id;
	int bi_in = id + (n/2);
	int ai = tid;
	int bi = tid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	int d;
	
	if (bi < n) {
		if (ai < (n/2)) {
        	        temp[ai + bankOffsetA] = g_idata[ai_in];
	        }
		temp[bi + bankOffsetB] = g_idata[bi_in];

		for (d = n >> 1; d > 32; d >>= 1) { 
			__syncthreads();
  			if (tid < d) {
				int ai = offset*(2*tid+1)-1;
				int bi = offset*(2*tid+2)-1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
	
				if (bi < n) {
					temp[bi] += temp[ai];
				}
                	}
			offset *= 2;
		}

  		if (tid < 32) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;
  		if (tid < 16) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;
  		if (tid < 8) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;
  		if (tid < 4) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;
  		if (tid < 2) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;
  		if (tid < 1) {
			int ai = offset*(2*tid+1)-1; int bi = offset*(2*tid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);	bi += CONFLICT_FREE_OFFSET(bi);
			if (bi < n) temp[bi] += temp[ai];
               	}
		offset *= 2;

		for (d = 1; d <= (n/2); d *= 2) {
     			offset >>= 1;
			int i = n >> 1, j = d*2;
			while (j <= i) {
				i >>= 1;
			}
	   		 __syncthreads();
   			 if (tid < i) {
     				 int ai = offset*(2*tid+2)-1;
     				 int bi = offset*(2*tid+3)-1;
	     			 ai += CONFLICT_FREE_OFFSET(ai);
      				 bi += CONFLICT_FREE_OFFSET(bi); 
				 if (bi < n) {
	        			 temp[bi] += temp[ai];
			 	}
	     		}
		}
		__syncthreads();

		if (ai < (n/2)) {
			g_odata[ai_in] = temp[ai + bankOffsetA];
		}
		if (bi < n) {
			g_odata[bi_in] = temp[bi + bankOffsetB];
		}
		if (tid == 0) {
                        sum[bid] = temp[n - 1 + CONFLICT_FREE_OFFSET(n-1)];
                }
	}
}

/* Input: F (Present Frontier), f_size (Frontier Size), AllocationSize (Offsets of starting neighbours in next frontier), R, C
 * Output: F_next (Next Frontier)
 * 
 * Serial Gathering of vertices for the next frontier
 * Each thread expands the neighbors of one vertex in F
 */
__global__ void getNextFrontier (const dtype* F, dtype* AllocationSize, dtype *R, dtype *C, dtype* F_next, int f_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
        dtype offset = 0;

        if (idx < f_size) {
                if (idx > 0) {
                        offset = AllocationSize[idx - 1];
                }
                dtype vertex = F[idx];
                dtype c_start = R[vertex];
                dtype c_end = R[vertex + 1];

                for (dtype i = c_start; i < c_end; i++) {
                        F_next[offset] = C[i];
                        offset++;
                }
        }
}

/* NOT USED PRESENTLY
 * Fine grained version of getNextFrontier: Serial gathering expands one vertex per thread which
 * results in unbalanced amounts of work done by each thread in a block. To balance this among
 * all the threads of a block, each thread tries to process FTS number of output elements. So, only
 * some threads of the block expand the required number of vertices and load the column indices into
 * shared memory. Each thread then uses these column indices to expand FTS vertices.
 * Binary search is used to compute the start vertex for each block to expand.
 */
__global__ void getNextFrontier2 (const dtype* F, dtype* AllocationSize, dtype *R, dtype *C, dtype* F_next, int f_size, int fnext_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int bid = blockIdx.x;
        int tid = threadIdx.x;
        __shared__ int Col[FTS * BLOCK_SIZE];
        dtype offset = 0;
        __shared__ int start;

        if (tid == 0) {
                int size = FTS * BLOCK_SIZE * bid;
                int low = 0, high = f_size - 1, mid;
                while (low < high) {
                        mid = (low + high) >> 1;
                        if (size <= AllocationSize[mid]) {
                                high = mid;
                        } else {
                                low = mid + 1;
                        }
                }
                start = (AllocationSize[high] == size) ? (high + 1) : high;
        }
        __syncthreads();
        if (start + tid < f_size) {
                offset = (start + tid > 0) ? AllocationSize[start + tid - 1] : 0;

                if (offset < FTS * BLOCK_SIZE * (bid + 1)) {
                        dtype vertex = F[start + tid];
                        int c_off = (tid == 0 && bid > 0) ? (FTS * BLOCK_SIZE * bid - offset) : 0;
                        dtype c_start = R[vertex] + c_off;
                        dtype c_end = R[vertex + 1];

                        for (int i = 0; (i < (c_end - c_start)) && (offset + i + c_off < FTS * BLOCK_SIZE * (bid + 1)); i++) {
                                Col[i + offset - FTS * BLOCK_SIZE * bid + c_off] = c_start + i;
                        }
                }
        }
        __syncthreads();

        for (int i = 0; i < FTS; i++) {
                if (idx * FTS + i < fnext_size) {
                        F_next[idx * FTS + i] = C[Col[tid * FTS + i]];
                }
        }
}

/* NOT USED PRESENTLY
 * Helper function for getNextFrontier3
 *
 * Compute the start indices in the next frontier that should be processed by each block in
 * getNextFrontier3
 */
__global__ void getStartIndices (dtype *AllocationSize, uint32_t f_size, uint32_t *sindices) {
        uint32_t count = 0;

        for (uint32_t i = 0; i < f_size; i++) {
                if (AllocationSize[i] > count * FTS * BLOCK_SIZE) {
                        sindices[count++] = i--;
                }
        }
}

/* NOT USED PRESENTLY
 * Another Fine grained version of getNextFrontier: Similar to getNextFrontier2, but uses precomputed start
 * indices for the blocks using getStartIndices instead of using the binary search.
 *
 * Each thread expands FTS elements of the next frontier.
 */
__global__ void getNextFrontier3 (const dtype* F, dtype* AllocationSize, dtype *R, dtype *C, dtype* F_next, uint32_t f_size, uint32_t fnext_size, uint32_t *sindices, uint32_t nBlocks) {
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t bid = blockIdx.x;
        uint32_t tid = threadIdx.x;
        __shared__ uint16_t Col[FTS * BLOCK_SIZE];
        dtype offset = 0;
        uint32_t start = sindices[bid];

        if ((start + tid) < f_size) {
                offset = (start + tid > 0) ? AllocationSize[start + tid - 1] : 0;

                if (offset < FTS * BLOCK_SIZE * (bid + 1)) {
                        dtype vertex = F[start + tid];
                        uint32_t c_off = (tid == 0 && bid > 0) ? (FTS * BLOCK_SIZE * bid - offset) : 0;
                        dtype c_start = R[vertex] + c_off;
                        dtype c_end = R[vertex + 1];
                        for (uint32_t i = 0; (i < (c_end - c_start)) && (offset + i + c_off < FTS * BLOCK_SIZE * (bid + 1)); i++) {
                                Col[i + offset - FTS * BLOCK_SIZE * bid + c_off] = c_start + i;
                        }
                }
        }
        __syncthreads();

        for (uint32_t i = 0; i < FTS; i++) {
                if (idx * FTS + i < fnext_size) {
                        F_next[idx * FTS + i] = C[Col[tid * FTS + i]];
                }
        }
}

/* Input: F (Present Frontier), fsize (Frontier Size), D (depth of vertices)
 * Output: mask (flags indicating if the vertex is newly discovered)
 * 
 * Check the depth of vertices gathered and set the mask to 0 or 1
 * This implicitly eliminates the duplicates as the duplicates would correspond to same index in mask
 * Each thread checks one vertex in F
 */
__global__ void generateMask (dtype *F, int fsize, int *mask, int *D) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < fsize) {
		dtype vertex = F[idx];
		mask[vertex] = (D[vertex] == -1) ? 1 : 0;
	}
}

/* Input: mask, mask_scan (Prefix sum of mask), mask_size
 * Output: F_compact (Compact frontier after filtering)
 *
 * Each thread processes one vertex or element of mask and sets the vertex number in F_compact
 * using the position obtained using mask_scan
 */
__global__ void compact (int *mask, int *mask_scan, dtype *F_compact, int mask_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < mask_size) {
		if (mask[idx] == 1) {
			if (idx == 0) {
				F_compact[0] = idx;
			} else {
				int index = mask_scan[idx - 1];
				F_compact[index] = idx;
			}
		}
	}
}

/*
 * Input: mask (Flag indicating newly discovered vertices), N (number of nodes)
 * Output: F_compact (Compact frontier), fcompact_size (Size of compact frontier)
 *
 * Sequential filtering of the frontier, when the size of filter is below the threshold
 * Single thread examines the mask and sets vertex numbers in F_compact
 */
__global__ void compactSeq (int *mask, int N, dtype *F_compact, int *fcompact_size) {
	int current = 0;
	for (int idx = 0; idx < N; idx++) {
		if (mask[idx] == 1) {
			F_compact[current++] = idx;
		}
	}
	*fcompact_size = current;
}

/*
 * Input: mask (Flags indicating newly discovered vertices), D (depth vector), N (number of nodes),
 * 	  depth (current level of BFS)
 * Output: d_done (flag indicating last iteration of BFS)
 *
 * Each thread checks the mask and sets the depth of the corr. vertex
 * If any of the vertices has an invalid depth, then sets d_done to false to indicate that the
 * search should continue to the next level.
 */
__global__ void setDepth (int* mask, int *D, int N, int depth, bool *d_done) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N) {
		if (mask[idx] == 1) {
			D[idx] = depth;
		} else if (D[idx] == -1) {
			*d_done = false;
		}
	}
}

/* The optimized BFS version based on the reference papaer by Duane Merill
 *
 * This does a frontier based search by using the CSR representation of the graph.
 * The search proceeds using various steps for gathering the neighbors of the vertices in the
 * present frontier and processing them to obtain unique newly discovered vertices.
 */
void
BFSMerill (dtype* C, dtype* R, unsigned int N, unsigned int num_edges, int* D)
{
	unsigned int nBlocks, nThreads, tbSize;
	dtype *AllocationSize, *F_next, *F_compact;
	dtype initial_F = 0;
	int f_size = 1, *d_fcompact_size, fcompact_size = 1, fnext_size;
	int current_depth = 0, mask_threshold = 512;
	int *mask, *mask_scan, *sum;
	bool h_true = true, h_done = false, *d_done = NULL;

        CUDA_CHECK_ERROR (hipMalloc ((void**) &d_done, sizeof (bool)));

	CUDA_CHECK_ERROR (hipMalloc ((void**) &F_compact, sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_fcompact_size, sizeof (int)));
	CUDA_CHECK_ERROR (hipMemcpy (F_compact, &initial_F, sizeof (dtype), hipMemcpyHostToDevice));

        while(!h_done) {
		CUDA_CHECK_ERROR (hipMemcpy(d_done, &h_true, sizeof(bool), hipMemcpyHostToDevice));
		current_depth++;

		/* Allocate memory required for vector of index offsets into next frontier */
		f_size = fcompact_size;
		CUDA_CHECK_ERROR (hipMalloc ((void**) &AllocationSize, f_size * sizeof (dtype)));

		/* Get index offsets into the next frontier for writing neighbors */
		nThreads = (f_size + 1) / 2;
		tbSize = BLOCK_SIZE;
		nBlocks = (nThreads + tbSize - 1) / tbSize;
		CUDA_CHECK_ERROR (hipMalloc ((void**) &sum, nBlocks * sizeof (int)));
                prefixscan <<<nBlocks, tbSize>>>((int *)AllocationSize, F_compact, f_size, tbSize, R, N, sum);

		sumPrefix <<<1,1>>> (sum, nBlocks);
		prefixscan2 <<<nBlocks, tbSize>>> ((int *)AllocationSize, f_size, tbSize, sum);
		CUDA_CHECK_ERROR (hipFree(sum));

		CUDA_CHECK_ERROR (hipMemcpy(&fnext_size, &AllocationSize[f_size-1], sizeof(int), hipMemcpyDeviceToHost));

		/* Allocate array for next frontier */
		CUDA_CHECK_ERROR (hipMalloc ((void**) &F_next, fnext_size * sizeof (dtype)));

		/* Gather the neighbors to form next frontier */
		nThreads = f_size;
                tbSize = BLOCK_SIZE;
                nBlocks = (nThreads + tbSize - 1) / tbSize;
		getNextFrontier <<<nBlocks, tbSize>>> (F_compact, AllocationSize, R, C, F_next, f_size);
		CUDA_CHECK_ERROR (hipFree (AllocationSize));
		CUDA_CHECK_ERROR (hipFree ((void*)F_compact));

		/* Mask to indicate newly discivered neighbors in the gathered frontier */
		CUDA_CHECK_ERROR (hipMalloc ((void**) &mask, N * sizeof (int)));
		CUDA_CHECK_ERROR (hipMemset(mask, 0, N * sizeof (int)));
			
		nThreads = fnext_size;
        	tbSize = BLOCK_SIZE;
                nBlocks = (nThreads + tbSize - 1) / tbSize;
		generateMask<<<nBlocks, tbSize>>> (F_next, fnext_size, mask, D);
		CUDA_CHECK_ERROR (hipFree (F_next));
		
		/* Set the depth of the new vertices at the current level of processing */
		nThreads = N;
		tbSize = BLOCK_SIZE;
		nBlocks = (nThreads + tbSize - 1) / tbSize;
		setDepth <<<nBlocks, tbSize>>> (mask, D, N, current_depth, d_done);
		
		if (fnext_size < mask_threshold) {
			/* Gather the compact frontier sequentially */
			CUDA_CHECK_ERROR (hipMalloc ((void**) &F_compact, fnext_size * sizeof (dtype)));

                        compactSeq <<<1, 1>>> (mask, N, F_compact, d_fcompact_size);
			CUDA_CHECK_ERROR (hipMemcpy(&fcompact_size, d_fcompact_size, sizeof(int), hipMemcpyDeviceToHost));
			if (!fcompact_size) {
				CUDA_CHECK_ERROR (hipFree (F_compact));
			}
		} else {
			CUDA_CHECK_ERROR (hipMalloc ((void**) &mask_scan, N * sizeof (int)));
			
			/* Prefix scan on mask to get the offsets into the compact frontier for next level*/
			nThreads = (N + 1)/2;
	                tbSize = BLOCK_SIZE;
        	        nBlocks = (nThreads + tbSize - 1) / tbSize;
			CUDA_CHECK_ERROR (hipMalloc ((void**) &sum, nBlocks * sizeof (int)));
			maskPrefixscan <<<nBlocks, tbSize>>> (mask_scan, mask, N, tbSize, sum);

			sumPrefix <<<1,1>>> (sum, nBlocks);
	                prefixscan2 <<<1, tbSize>>> (mask_scan, N, tbSize, sum);
			CUDA_CHECK_ERROR (hipFree (sum));
			CUDA_CHECK_ERROR (hipMemcpy(&fcompact_size, &mask_scan[N-1], sizeof(int), hipMemcpyDeviceToHost));

			if (fcompact_size) {
				CUDA_CHECK_ERROR (hipMalloc ((void**) &F_compact, fcompact_size * sizeof (dtype)));

				/* Kernel to compact the frontier after filtering */
				nThreads = N;
        	        	tbSize = BLOCK_SIZE;
                		nBlocks = (nThreads + tbSize - 1) / tbSize;
				compact <<<nBlocks, tbSize>>> (mask, mask_scan, F_compact, N);
			}
			CUDA_CHECK_ERROR (hipFree (mask));
			CUDA_CHECK_ERROR (hipFree (mask_scan));
		}
		CUDA_CHECK_ERROR (hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost));
        }
	CUDA_CHECK_ERROR (hipFree (d_fcompact_size));
	CUDA_CHECK_ERROR (hipFree (d_done));
}

/* Host function for calling the optimized BFS version based on the reference paper by Duane Merill.*/
void
cudaBFS_Merill (dtype* h_C, dtype* h_R, unsigned int N, unsigned int num_edges, int* h_D)
{
	dtype *d_C, *d_R;
	int *d_D;
	
	struct timeval start, end;
	gettimeofday(&start, NULL);

	h_D[0] = 0;
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_C, num_edges * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_C, h_C, num_edges * sizeof (dtype), hipMemcpyHostToDevice));

	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_R, (N + 1) * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_R, h_R, (N + 1) * sizeof (dtype), hipMemcpyHostToDevice));

	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_D, N * sizeof (int)));
	CUDA_CHECK_ERROR (hipMemcpy (d_D, h_D, N * sizeof (int), hipMemcpyHostToDevice));
	
	BFSMerill (d_C, d_R, N, num_edges, d_D);

	gettimeofday(&end, NULL);

	CUDA_CHECK_ERROR (hipMemcpy (h_D, d_D, N * sizeof (int), hipMemcpyDeviceToHost));

	fprintf (stderr, "Time to execute GPU BFS Merill code: %ld usecs\n", ((end.tv_sec * 1000000 + end.tv_usec)
										  - (start.tv_sec * 1000000 + start.tv_usec)));
	CUDA_CHECK_ERROR (hipFree (d_C));
	CUDA_CHECK_ERROR (hipFree (d_R));
	CUDA_CHECK_ERROR (hipFree (d_D));
}
